#include "hip/hip_runtime.h"
// Licensed to the Apache Software Foundation (ASF) under one
// or more contributor license agreements.  See the NOTICE file
// distributed with this work for additional information
// regarding copyright ownership.  The ASF licenses this file
// to you under the Apache License, Version 2.0 (the
// "License"); you may not use this file except in compliance
// with the License.  You may obtain a copy of the License at
//
//   http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing,
// software distributed under the License is distributed on an
// "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
// KIND, either express or implied.  See the License for the
// specific language governing permissions and limitations
// under the License.


#include "faiss/FaissAssert.h"
#include "faiss/gpu/utils/Limits.cuh"
#include "Arithmetic.h"


namespace faiss {
namespace gpu {

constexpr bool kBoolMax = zilliz::milvus::engine::kBoolMax;
constexpr bool kBoolMin = zilliz::milvus::engine::kBoolMin;

template<>
struct Limits<bool> {
    static __device__ __host__
    inline bool getMin() {
        return kBoolMin;
    }
    static __device__ __host__
    inline bool getMax() {
        return kBoolMax;
    }
};

constexpr int8_t kInt8Max = zilliz::milvus::engine::kInt8Max;
constexpr int8_t kInt8Min = zilliz::milvus::engine::kInt8Min;

template<>
struct Limits<int8_t> {
    static __device__ __host__
    inline int8_t getMin() {
        return kInt8Min;
    }
    static __device__ __host__
    inline int8_t getMax() {
        return kInt8Max;
    }
};

constexpr int16_t kInt16Max = zilliz::milvus::engine::kInt16Max;
constexpr int16_t kInt16Min = zilliz::milvus::engine::kInt16Min;

template<>
struct Limits<int16_t> {
    static __device__ __host__
    inline int16_t getMin() {
        return kInt16Min;
    }
    static __device__ __host__
    inline int16_t getMax() {
        return kInt16Max;
    }
};

constexpr int64_t kInt64Max = zilliz::milvus::engine::kInt64Max;
constexpr int64_t kInt64Min = zilliz::milvus::engine::kInt64Min;

template<>
struct Limits<int64_t> {
    static __device__ __host__
    inline int64_t getMin() {
        return kInt64Min;
    }
    static __device__ __host__
    inline int64_t getMax() {
        return kInt64Max;
    }
};

constexpr double kDoubleMax = zilliz::milvus::engine::kDoubleMax;
constexpr double kDoubleMin = zilliz::milvus::engine::kDoubleMin;

template<>
struct Limits<double> {
    static __device__ __host__
    inline double getMin() {
        return kDoubleMin;
    }
    static __device__ __host__
    inline double getMax() {
        return kDoubleMax;
    }
};

}
}

#include "faiss/gpu/utils/DeviceUtils.h"
#include "faiss/gpu/utils/MathOperators.cuh"
#include "faiss/gpu/utils/Pair.cuh"
#include "faiss/gpu/utils/Reductions.cuh"
#include "faiss/gpu/utils/Select.cuh"
#include "faiss/gpu/utils/Tensor.cuh"
#include "faiss/gpu/utils/StaticUtils.h"

#include "Topk.h"


namespace zilliz {
namespace milvus {
namespace engine {
namespace gpu {

constexpr int kWarpSize = 32;

template<typename T, int Dim, bool InnerContig>
using Tensor = faiss::gpu::Tensor<T, Dim, InnerContig>;

template<typename T, typename U>
using Pair = faiss::gpu::Pair<T, U>;


// select kernel for k == 1
template<typename T, int kRowsPerBlock, int kBlockSize>
__global__ void topkSelectMin1(Tensor<T, 2, true> productDistances,
                               Tensor<T, 2, true> outDistances,
                               Tensor<int64_t, 2, true> outIndices) {
    // Each block handles kRowsPerBlock rows of the distances (results)
    Pair<T, int64_t> threadMin[kRowsPerBlock];
    __shared__
    Pair<T, int64_t> blockMin[kRowsPerBlock * (kBlockSize / kWarpSize)];

    T distance[kRowsPerBlock];

#pragma unroll
    for (int i = 0; i < kRowsPerBlock; ++i) {
        threadMin[i].k = faiss::gpu::Limits<T>::getMax();
        threadMin[i].v = -1;
    }

    // blockIdx.x: which chunk of rows we are responsible for updating
    int rowStart = blockIdx.x * kRowsPerBlock;

    // FIXME: if we have exact multiples, don't need this
    bool endRow = (blockIdx.x == gridDim.x - 1);

    if (endRow) {
        if (productDistances.getSize(0) % kRowsPerBlock == 0) {
            endRow = false;
        }
    }

    if (endRow) {
        for (int row = rowStart; row < productDistances.getSize(0); ++row) {
            for (int col = threadIdx.x; col < productDistances.getSize(1);
                 col += blockDim.x) {
                distance[0] = productDistances[row][col];

                if (faiss::gpu::Math<T>::lt(distance[0], threadMin[0].k)) {
                    threadMin[0].k = distance[0];
                    threadMin[0].v = col;
                }
            }

            // Reduce within the block
            threadMin[0] =
                faiss::gpu::blockReduceAll<Pair<T, int64_t>, faiss::gpu::Min<Pair<T, int64_t> >, false, false>(
                    threadMin[0], faiss::gpu::Min<Pair<T, int64_t> >(), blockMin);

            if (threadIdx.x == 0) {
                outDistances[row][0] = threadMin[0].k;
                outIndices[row][0] = threadMin[0].v;
            }

            // so we can use the shared memory again
            __syncthreads();

            threadMin[0].k = faiss::gpu::Limits<T>::getMax();
            threadMin[0].v = -1;
        }
    } else {
        for (int col = threadIdx.x; col < productDistances.getSize(1);
             col += blockDim.x) {

#pragma unroll
            for (int row = 0; row < kRowsPerBlock; ++row) {
                distance[row] = productDistances[rowStart + row][col];
            }

#pragma unroll
            for (int row = 0; row < kRowsPerBlock; ++row) {
                if (faiss::gpu::Math<T>::lt(distance[row], threadMin[row].k)) {
                    threadMin[row].k = distance[row];
                    threadMin[row].v = col;
                }
            }
        }

        // Reduce within the block
        faiss::gpu::blockReduceAll<kRowsPerBlock, Pair<T, int64_t>, faiss::gpu::Min<Pair<T, int64_t> >, false, false>(
            threadMin, faiss::gpu::Min<Pair<T, int64_t> >(), blockMin);

        if (threadIdx.x == 0) {
#pragma unroll
            for (int row = 0; row < kRowsPerBlock; ++row) {
                outDistances[rowStart + row][0] = threadMin[row].k;
                outIndices[rowStart + row][0] = threadMin[row].v;
            }
        }
    }
}

// L2 + select kernel for k > 1, no re-use of ||c||^2
template<typename T, int NumWarpQ, int NumThreadQ, int ThreadsPerBlock>
__global__ void topkSelectMinK(Tensor<T, 2, true> productDistances,
                               Tensor<T, 2, true> outDistances,
                               Tensor<int64_t, 2, true> outIndices,
                               int k, T initK) {
    // Each block handles a single row of the distances (results)
    constexpr int kNumWarps = ThreadsPerBlock / kWarpSize;

    __shared__
    T smemK[kNumWarps * NumWarpQ];
    __shared__
    int64_t smemV[kNumWarps * NumWarpQ];

    faiss::gpu::BlockSelect<T, int64_t, false, faiss::gpu::Comparator<T>,
                            NumWarpQ, NumThreadQ, ThreadsPerBlock>
        heap(initK, -1, smemK, smemV, k);

    int row = blockIdx.x;

    // Whole warps must participate in the selection
    int limit = faiss::gpu::utils::roundDown(productDistances.getSize(1), kWarpSize);
    int i = threadIdx.x;

    for (; i < limit; i += blockDim.x) {
        T v = productDistances[row][i];
        heap.add(v, i);
    }

    if (i < productDistances.getSize(1)) {
        T v = productDistances[row][i];
        heap.addThreadQ(v, i);
    }

    heap.reduce();
    for (int i = threadIdx.x; i < k; i += blockDim.x) {
        outDistances[row][i] = smemK[i];
        outIndices[row][i] = smemV[i];
    }
}

// FIXME: no TVec specialization
template<typename T>
void runTopKSelectMin(Tensor<T, 2, true> &productDistances,
                      Tensor<T, 2, true> &outDistances,
                      Tensor<int64_t, 2, true> &outIndices,
                      int k,
                      hipStream_t stream) {
    FAISS_ASSERT(productDistances.getSize(0) == outDistances.getSize(0));
    FAISS_ASSERT(productDistances.getSize(0) == outIndices.getSize(0));
    FAISS_ASSERT(outDistances.getSize(1) == k);
    FAISS_ASSERT(outIndices.getSize(1) == k);
    FAISS_ASSERT(k <= 1024);

    if (k == 1) {
        constexpr int kThreadsPerBlock = 256;
        constexpr int kRowsPerBlock = 8;

        auto block = dim3(kThreadsPerBlock);
        auto grid = dim3(faiss::gpu::utils::divUp(outDistances.getSize(0), kRowsPerBlock));

        topkSelectMin1<T, kRowsPerBlock, kThreadsPerBlock>
            << < grid, block, 0, stream >> > (productDistances, outDistances, outIndices);
    } else {
        constexpr int kThreadsPerBlock = 128;

        auto block = dim3(kThreadsPerBlock);
        auto grid = dim3(outDistances.getSize(0));

#define RUN_TOPK_SELECT_MIN(NUM_WARP_Q, NUM_THREAD_Q)                         \
    do {                                                                \
      topkSelectMinK<T, NUM_WARP_Q, NUM_THREAD_Q, kThreadsPerBlock>       \
        <<<grid, block, 0, stream>>>(productDistances, \
                                     outDistances, outIndices,          \
                                     k, faiss::gpu::Limits<T>::getMax());           \
    } while (0)

        if (k <= 32) {
            RUN_TOPK_SELECT_MIN(32, 2);
        } else if (k <= 64) {
            RUN_TOPK_SELECT_MIN(64, 3);
        } else if (k <= 128) {
            RUN_TOPK_SELECT_MIN(128, 3);
        } else if (k <= 256) {
            RUN_TOPK_SELECT_MIN(256, 4);
        } else if (k <= 512) {
            RUN_TOPK_SELECT_MIN(512, 8);
        } else if (k <= 1024) {
            RUN_TOPK_SELECT_MIN(1024, 8);
        } else {
            FAISS_ASSERT(false);
        }
    }

    CUDA_TEST_ERROR();
}

////////////////////////////////////////////////////////////
// select kernel for k == 1
template<typename T, int kRowsPerBlock, int kBlockSize>
__global__ void topkSelectMax1(Tensor<T, 2, true> productDistances,
                               Tensor<T, 2, true> outDistances,
                               Tensor<int64_t, 2, true> outIndices) {
    // Each block handles kRowsPerBlock rows of the distances (results)
    Pair<T, int64_t> threadMax[kRowsPerBlock];
    __shared__
    Pair<T, int64_t> blockMax[kRowsPerBlock * (kBlockSize / kWarpSize)];

    T distance[kRowsPerBlock];

#pragma unroll
    for (int i = 0; i < kRowsPerBlock; ++i) {
        threadMax[i].k = faiss::gpu::Limits<T>::getMin();
        threadMax[i].v = -1;
    }

    // blockIdx.x: which chunk of rows we are responsible for updating
    int rowStart = blockIdx.x * kRowsPerBlock;

    // FIXME: if we have exact multiples, don't need this
    bool endRow = (blockIdx.x == gridDim.x - 1);

    if (endRow) {
        if (productDistances.getSize(0) % kRowsPerBlock == 0) {
            endRow = false;
        }
    }

    if (endRow) {
        for (int row = rowStart; row < productDistances.getSize(0); ++row) {
            for (int col = threadIdx.x; col < productDistances.getSize(1);
                 col += blockDim.x) {
                distance[0] = productDistances[row][col];

                if (faiss::gpu::Math<T>::gt(distance[0], threadMax[0].k)) {
                    threadMax[0].k = distance[0];
                    threadMax[0].v = col;
                }
            }

            // Reduce within the block
            threadMax[0] =
                faiss::gpu::blockReduceAll<Pair<T, int64_t>, faiss::gpu::Max<Pair<T, int64_t> >, false, false>(
                    threadMax[0], faiss::gpu::Max<Pair<T, int64_t> >(), blockMax);

            if (threadIdx.x == 0) {
                outDistances[row][0] = threadMax[0].k;
                outIndices[row][0] = threadMax[0].v;
            }

            // so we can use the shared memory again
            __syncthreads();

            threadMax[0].k = faiss::gpu::Limits<T>::getMin();
            threadMax[0].v = -1;
        }
    } else {
        for (int col = threadIdx.x; col < productDistances.getSize(1);
             col += blockDim.x) {

#pragma unroll
            for (int row = 0; row < kRowsPerBlock; ++row) {
                distance[row] = productDistances[rowStart + row][col];
            }

#pragma unroll
            for (int row = 0; row < kRowsPerBlock; ++row) {
                if (faiss::gpu::Math<T>::gt(distance[row], threadMax[row].k)) {
                    threadMax[row].k = distance[row];
                    threadMax[row].v = col;
                }
            }
        }

        // Reduce within the block
        faiss::gpu::blockReduceAll<kRowsPerBlock, Pair<T, int64_t>, faiss::gpu::Max<Pair<T, int64_t> >, false, false>(
            threadMax, faiss::gpu::Max<Pair<T, int64_t> >(), blockMax);

        if (threadIdx.x == 0) {
#pragma unroll
            for (int row = 0; row < kRowsPerBlock; ++row) {
                outDistances[rowStart + row][0] = threadMax[row].k;
                outIndices[rowStart + row][0] = threadMax[row].v;
            }
        }
    }
}

// L2 + select kernel for k > 1, no re-use of ||c||^2
template<typename T, int NumWarpQ, int NumThreadQ, int ThreadsPerBlock>
__global__ void topkSelectMaxK(Tensor<T, 2, true> productDistances,
                               Tensor<T, 2, true> outDistances,
                               Tensor<int64_t, 2, true> outIndices,
                               int k, T initK) {
    // Each block handles a single row of the distances (results)
    constexpr int kNumWarps = ThreadsPerBlock / kWarpSize;

    __shared__
    T smemK[kNumWarps * NumWarpQ];
    __shared__
    int64_t smemV[kNumWarps * NumWarpQ];

    faiss::gpu::BlockSelect<T, int64_t, true, faiss::gpu::Comparator<T>,
                            NumWarpQ, NumThreadQ, ThreadsPerBlock>
        heap(initK, -1, smemK, smemV, k);

    int row = blockIdx.x;

    // Whole warps must participate in the selection
    int limit = faiss::gpu::utils::roundDown(productDistances.getSize(1), kWarpSize);
    int i = threadIdx.x;

    for (; i < limit; i += blockDim.x) {
        T v = productDistances[row][i];
        heap.add(v, i);
    }

    if (i < productDistances.getSize(1)) {
        T v = productDistances[row][i];
        heap.addThreadQ(v, i);
    }

    heap.reduce();
    for (int i = threadIdx.x; i < k; i += blockDim.x) {
        outDistances[row][i] = smemK[i];
        outIndices[row][i] = smemV[i];
    }
}

// FIXME: no TVec specialization
template<typename T>
void runTopKSelectMax(Tensor<T, 2, true> &productDistances,
                      Tensor<T, 2, true> &outDistances,
                      Tensor<int64_t, 2, true> &outIndices,
                      int k,
                      hipStream_t stream) {
    FAISS_ASSERT(productDistances.getSize(0) == outDistances.getSize(0));
    FAISS_ASSERT(productDistances.getSize(0) == outIndices.getSize(0));
    FAISS_ASSERT(outDistances.getSize(1) == k);
    FAISS_ASSERT(outIndices.getSize(1) == k);
    FAISS_ASSERT(k <= 1024);

    if (k == 1) {
        constexpr int kThreadsPerBlock = 256;
        constexpr int kRowsPerBlock = 8;

        auto block = dim3(kThreadsPerBlock);
        auto grid = dim3(faiss::gpu::utils::divUp(outDistances.getSize(0), kRowsPerBlock));

        topkSelectMax1<T, kRowsPerBlock, kThreadsPerBlock>
            << < grid, block, 0, stream >> > (productDistances, outDistances, outIndices);
    } else {
        constexpr int kThreadsPerBlock = 128;

        auto block = dim3(kThreadsPerBlock);
        auto grid = dim3(outDistances.getSize(0));

#define RUN_TOPK_SELECT_MAX(NUM_WARP_Q, NUM_THREAD_Q)                         \
    do {                                                                \
      topkSelectMaxK<T, NUM_WARP_Q, NUM_THREAD_Q, kThreadsPerBlock>       \
        <<<grid, block, 0, stream>>>(productDistances, \
                                     outDistances, outIndices,          \
                                     k, faiss::gpu::Limits<T>::getMin());           \
    } while (0)

        if (k <= 32) {
            RUN_TOPK_SELECT_MAX(32, 2);
        } else if (k <= 64) {
            RUN_TOPK_SELECT_MAX(64, 3);
        } else if (k <= 128) {
            RUN_TOPK_SELECT_MAX(128, 3);
        } else if (k <= 256) {
            RUN_TOPK_SELECT_MAX(256, 4);
        } else if (k <= 512) {
            RUN_TOPK_SELECT_MAX(512, 8);
        } else if (k <= 1024) {
            RUN_TOPK_SELECT_MAX(1024, 8);
        } else {
            FAISS_ASSERT(false);
        }
    }

    CUDA_TEST_ERROR();
}
//////////////////////////////////////////////////////////////

template<typename T>
void runTopKSelect(Tensor<T, 2, true> &productDistances,
                   Tensor<T, 2, true> &outDistances,
                   Tensor<int64_t, 2, true> &outIndices,
                   bool dir,
                   int k,
                   hipStream_t stream) {
    if (dir) {
        runTopKSelectMax<T>(productDistances,
                            outDistances,
                            outIndices,
                            k,
                            stream);
    } else {
        runTopKSelectMin<T>(productDistances,
                            outDistances,
                            outIndices,
                            k,
                            stream);
    }
}

template<typename T>
void TopK(T *input,
          int length,
          int k,
          T *output,
          int64_t *idx,
//          Ordering order_flag,
          hipStream_t stream) {

//    bool dir = (order_flag == Ordering::kAscending ? false : true);
    bool dir = 0;

    Tensor<T, 2, true> t_input(input, {1, length});
    Tensor<T, 2, true> t_output(output, {1, k});
    Tensor<int64_t, 2, true> t_idx(idx, {1, k});

    runTopKSelect<T>(t_input, t_output, t_idx, dir, k, stream);
}

//INSTANTIATION_TOPK_2(bool);
//INSTANTIATION_TOPK_2(int8_t);
//INSTANTIATION_TOPK_2(int16_t);
INSTANTIATION_TOPK_2(int32_t);
//INSTANTIATION_TOPK_2(int64_t);
INSTANTIATION_TOPK_2(float);
//INSTANTIATION_TOPK_2(double);
//INSTANTIATION_TOPK(TimeInterval);
//INSTANTIATION_TOPK(Float128);
//INSTANTIATION_TOPK(char);

}

void TopK(float *host_input,
          int length,
          int k,
          float *output,
          int64_t *indices) {
    float *device_input, *device_output;
    int64_t *ids;

    hipMalloc((void **) &device_input, sizeof(float) * length);
    hipMalloc((void **) &device_output, sizeof(float) * k);
    hipMalloc((void **) &ids, sizeof(int64_t) * k);

    hipMemcpy(device_input, host_input, sizeof(float) * length, hipMemcpyHostToDevice);

    gpu::TopK<float>(device_input, length, k, device_output, ids, nullptr);

    hipMemcpy(output, device_output, sizeof(float) * k, hipMemcpyDeviceToHost);
    hipMemcpy(indices, ids, sizeof(int64_t) * k, hipMemcpyDeviceToHost);

    hipFree(device_input);
    hipFree(device_output);
    hipFree(ids);
}

}
}
}
